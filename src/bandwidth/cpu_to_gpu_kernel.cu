#include "hip/hip_runtime.h"
#include <iostream>
#include <lyra/lyra.hpp>

#include "common/cache_control.hpp"
#include "common/check_cuda.cuh"
#include "common/init.hpp"
#include "common/logger.hpp"
#include "common/perf_control.hpp"
#include "common/test_system_allocator.hpp"

template <typename T>
__forceinline__ __device__ void copy_func(char *__restrict__ dst,
                                          const char *__restrict__ src,
                                          const size_t n) {
  const size_t nElems = n / sizeof(T);

  auto dstP = reinterpret_cast<T *>(dst);
  auto srcP = reinterpret_cast<const T *>(src);

  for (size_t i = 0; i < nElems; i += blockDim.x * gridDim.x) {
    dstP[i] = srcP[i];
  }
}

template <typename T>
__global__ void copy_kernel(char *__restrict__ dst,
                            const char *__restrict__ src, const size_t n) {
  // copy in blocks of sizeof(T)
  copy_func<T>(dst, src, n);

  // copy in blocks of 1
  size_t rem = n - (n / sizeof(T));
  char *dstTail = &dst[n - rem];
  const char *srcTail = &src[n - rem];
  copy_func<char>(dstTail, srcTail, n);
}

int main(int argc, char **argv) {
  init();

  bool help = false;
  bool debug = false;
  bool verbose = false;
  bool noAtsCheck = false;
  bool strictPerf = false;
  bool flush = false;
  size_t n = 0;

  auto cli =
      lyra::help(help) |
      lyra::opt(debug)["--debug"]("print debug messages to stderr") |
      lyra::opt(verbose)["--verbose"]("print verbose messages to stderr") |
      lyra::opt(noAtsCheck)["--no-ats-check"]("skip test for ats") |
      lyra::opt(flush)["--flush"]("flush CPU cache") |
      lyra::opt(strictPerf)["--strict-perf"](
          "fail if system performance cannot be controlled") |
      lyra::arg(n, "size")("Size").required();

  auto result = cli.parse({argc, argv});
  if (!result) {
    LOG(error, "Error in command line: {}", result.errorMessage());
    exit(1);
  }

  if (help) {
    std::cout << cli;
    return 0;
  }

  // set logging level
  if (verbose) {
    logger::set_level(logger::Level::TRACE);
  } else if (debug) {
    logger::set_level(logger::Level::DEBUG);
  } else {
    logger::set_level(logger::Level::INFO);
  }

  // log command line before much else happens
  {
    std::string cmd;
    for (int i = 0; i < argc; ++i) {
      if (i != 0) {
        cmd += " ";
      }
      cmd += argv[i];
    }
    LOG(debug, cmd);
  }

  // test system allocator before any CUDA
  if (!noAtsCheck) {
    if (test_system_allocator()) {
      LOG(info, "CUDA supports system allocator");
    } else {
      LOG(critical, "CUDA does not work with the system allocator");
      exit(EXIT_FAILURE);
    }
  }

  // set CPU to high performance mode
  WithPerformance performanceGovernor(strictPerf);

  // disable CPU boosting
  WithoutBoost boostDisabler(strictPerf);

  typedef int32_t Type;
  const size_t nElems = n;
  const size_t nBytes = nElems * sizeof(Type);
  char *dst;
  CUDA_RUNTIME(hipMalloc(&dst, nBytes));
  char *src = new char[n * sizeof(Type)];
  if (!src) {
    LOG(critical, "failed allocation");
    exit(EXIT_FAILURE);
  }

  // touch all src lines
  LOG(info, "CPU touch src allocation");
  const size_t lineSize = cache_linesize();
  LOG(debug, "CPU line size {}", lineSize);
  for (size_t i = 0; i < nElems; i += lineSize / sizeof(Type)) {
    src[i] = 0;
  }

  // flush src pages from cache
  if (flush) {
    LOG(info, "flush CPU cache");
    flush_all(src, nBytes);
  }

  // create stream
  std::vector<hipStream_t> streams(1);
  CUDA_RUNTIME(hipStreamCreate(&streams[0]));

  // create event
  hipEvent_t start, stop;
  CUDA_RUNTIME(hipEventCreate(&start));
  CUDA_RUNTIME(hipEventCreate(&stop));

  // copy to GPU
  LOG(info, "operation");
  CUDA_RUNTIME(hipEventRecord(start, streams[0]));
  copy_kernel<uint32_t><<<250, 512, 0, streams[0]>>>(dst, src, nBytes);
  CUDA_RUNTIME(hipEventRecord(stop, streams[0]));

  // wait for copy to be done
  CUDA_RUNTIME(hipEventSynchronize(stop));

  float millis;
  CUDA_RUNTIME(hipEventElapsedTime(&millis, start, stop));
  double bytesPerSec = (nBytes / millis) * 1e3;
  fmt::print("{} {} {}\n", nBytes, bytesPerSec, millis / 1e3);

  // destroy stream
  for (auto stream : streams) {
    CUDA_RUNTIME(hipStreamDestroy(stream));
  }

  return 0;
}